#include "hip/hip_runtime.h"
#include "count_instrs.h"

#include <cstdint>
#include <utils/utils.h>

#include <hip/hip_runtime.h>

namespace device::count_instr {

extern "C" __device__ __noinline__ void IMPL_DETAIL_COUNT_INSTR_KERNEL(int predicate, int count_warp_level, uint64_t pcounter) {
    const int active_mask = __ballot_sync(__activemask(), 1); /* all the active threads will compute the active mask */
    const int laneid = get_laneid(); /* each thread will get a lane id (get_lane_id is implemented in utils/utils.h) */
    const int first_laneid = __ffs(active_mask) - 1; /* get the id of the first active thread */
    if (first_laneid != laneid) {  /* only the first active thread will perform the atomic */
        return;
    }

    const int predicate_mask = __ballot_sync(__activemask(), predicate); /* compute the predicate mask */
    const int num_threads = __popc(predicate_mask); /* count all the active threads */
    if (count_warp_level) {
        if (num_threads > 0) { /* num threads can be zero when accounting for predicates off */
            atomicAdd(reinterpret_cast<unsigned long long*>(pcounter), 1);
        }
    } else {
        atomicAdd(reinterpret_cast<unsigned long long*>(pcounter), num_threads);
    }
}

}