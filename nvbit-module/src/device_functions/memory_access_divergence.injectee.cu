#include "hip/hip_runtime.h"
// Implementation based on Listing 8 from the paper: "Oreste Villa, Mark Stephenson, David Nellans, and Stephen W. Keckler. 2019. NVBit: A
// Dynamic Binary Instrumentation Framework for NVIDIA GPUs. In Proceedings of the 52nd Annual IEEE/ACM International Symposium on
// Microarchitecture< (MICRO '52). Association for Computing Machinery, New York, NY, USA, 372–383.
// DOI:https://doi.org/10.1145/3352460.3358307"

#include "device_utility_functions.h"
#include "memory_access_divergence.h"

#include <cstdint>
#include <hip/hip_runtime.h>

namespace device::memory_access_divergence {

// __match_any_sync() is supported for CC 7.0 and higher
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#warp-match-functions
#if __CUDA_ARCH__ >= 700

extern "C" __device__ __noinline__ void IMPL_DETAIL_MEM_ACCESS_DIVERGENCE_KERNEL(
		int predicate, uint64_t address, uint64_t uniqueCacheLinesAccesses, uint64_t memoryAccessesCount) {
	if (!predicate) {
		return;
	}

	if (util::isFirstActiveThread()) {
		atomicAdd(reinterpret_cast<int*>(memoryAccessesCount), 1);
	}

	constexpr auto cacheLineSize{7}; // log2(128)
	const auto cacheLineAddress{address >> cacheLineSize};

	const auto activeMask{__ballot_sync(__activemask(), 1)};
	const auto threadsAccessingCacheLine{__popc(__match_any_sync(activeMask, cacheLineAddress))};
	// each thread contributes proportionally to the cache line counter, see
	// https://github.com/NVlabs/NVBit/issues/24#issuecomment-661176067
	atomicAdd(reinterpret_cast<float*>(uniqueCacheLinesAccesses), 1.f / threadsAccessingCacheLine);
}

#endif // __CUDA_ARCH__ >= 700

} // namespace device::memory_access_divergence