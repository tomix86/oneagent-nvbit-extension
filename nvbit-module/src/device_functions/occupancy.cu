#include "Configuration.h"
#include "Logger.h"
#include "communication/InstrumentationId.h"
#include "communication/MeasurementsPublisher.h"
#include "occupancy.h"
#include "util/cuda_utilities.h"

#include <hip/hip_runtime.h>

namespace device::occupancy {

void instrumentKernel(
		hipCtx_t context,
		int is_exit,
		nvbit_api_cuda_t /*eventId*/,
		cuLaunch_params* params,
		communication::MeasurementsPublisher& measurementsPublisher) {
	if (is_exit) {
		return;
	}

	const auto kernelName = nvbit_get_func_name(context, params->f, config::get().mangled ? 1 : 0);

	logging::info("Instrumenting kernel \"{}\" with occupancy calculation", kernelName);

	int device{};
	checkCudaErrors(hipGetDevice(&device));

	int maxBlocks{};
	checkCudaErrors(hipDeviceGetAttribute(&maxBlocks, hipDeviceAttributeMaxBlocksPerMultiprocessor, device));

	// Documentation for cuLaunchKernel_params is available here:
	// https://docs.nvidia.com/cuda/cuda-driver-api/group__CUDA__EXEC.html#group__CUDA__EXEC_1gb8f3dc3031b40da29d5f9a7139e52e15
	const auto kernelLaunchParams = reinterpret_cast<cuLaunchKernel_params*>(params);
	const auto num_ctas = kernelLaunchParams->gridDimX * kernelLaunchParams->gridDimY * kernelLaunchParams->gridDimZ;

	int numBlocks{};
	checkCudaErrors(hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, params->f, num_ctas, kernelLaunchParams->sharedMemBytes));
	const auto occupancy{100 * numBlocks / maxBlocks};

	logging::info("kernel \"{}\" occupancy: {}% ({} / {})", kernelName, occupancy, numBlocks, maxBlocks);
	measurementsPublisher.publish(communication::InstrumentationId::occupancy, std::to_string(occupancy));
}

} // namespace device::occupancy