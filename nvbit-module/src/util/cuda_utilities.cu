#include "Logger.h"
#include "cuda_utilities.h"

namespace util {

void checkError(hipError_t result, const char* calledFunc, std::string file, int line) {
	if (result == hipSuccess) {
		return;
	}

	const auto relativeFilePath{file.substr(file.rfind("src/"))};
	logging::warning("{} failed ({}:{}) code {} ({})", calledFunc, relativeFilePath, line, result, hipGetErrorString(result));
}

void checkError(hipError_t result, const char* calledFunc, std::string file, int line) {
	if (result == hipSuccess) {
		return;
	}

	const auto relativeFilePath{file.substr(file.rfind("src/"))};

	const char* errorString{};
	hipDrvGetErrorString(result, &errorString);

	logging::warning(
			"{} failed ({}:{}) code {} ({})",
			calledFunc,
			relativeFilePath,
			line,
			result,
			errorString ? errorString : "failed to retrieve error string");
}

} // namespace util