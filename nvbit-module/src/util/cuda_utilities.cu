#include "Logger.h"
#include "cuda_utilities.h"

#include <fmt/core.h>

namespace util {

void checkError(hipError_t result, const char* calledFunc, std::string file, int line) {
	if (result == hipSuccess) {
		return;
	}

	const auto relativeFilePath{file.substr(file.rfind("src/"))};
	logging::warning("{} failed ({}:{}) code {} ({})", calledFunc, relativeFilePath, line, result, hipGetErrorString(result));
}

void checkError(hipError_t result, const char* calledFunc, std::string file, int line) {
	if (result == hipSuccess) {
		return;
	}

	const auto relativeFilePath{file.substr(file.rfind("src/"))};

	const char* errorString{};
	hipDrvGetErrorString(result, &errorString);

	logging::warning(
			"{} failed ({}:{}) code {} ({})",
			calledFunc,
			relativeFilePath,
			line,
			result,
			errorString ? errorString : "failed to retrieve error string");
}

bool InstrumentationCache::isInstrumented(const hipFunction_t& function) {
	return !already_instrumented.insert(function).second;
}

ComputeCapability getComputeCapability() {
	int device;
	hipGetDevice(&device);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);
	return {deviceProp.major, deviceProp.minor};
}

std::string ComputeCapability::toString() const {
	return fmt::format("{},{}", major, minor);
}

} // namespace util