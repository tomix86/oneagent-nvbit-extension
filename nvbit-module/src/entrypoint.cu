#include "Configuration.h"
#include "Logger.h"
#include "communication/MeasurementsPublisher.h"
#include "communication/RuntimeConfigurationPoller.h"
#include "device_functions/branch_divergence.h"
#include "device_functions/count_instrs.h"
#include "device_functions/gmem_access_coalescence.h"
#include "device_functions/occupancy.h"

#include <boost/algorithm/cxx11/any_of.hpp>
#include <nvbit_tool.h> // Must be included only once!

communication::RuntimeConfigurationPoller runtimeConfigPoller;
communication::MeasurementsPublisher measurementsPublisher;

static void instrumentKernelLaunch(
		hipCtx_t context,
		int is_exit,
		nvbit_api_cuda_t eventId,
		cuLaunch_params* params,
		const std::vector<communication::InstrumentationId>& instrumentationFunctions) {
	for (const auto& functionId : instrumentationFunctions) {
		switch (functionId) {
		case communication::InstrumentationId::instructions_count:
			device::count_instr::instrumentKernel(context, is_exit, eventId, params, measurementsPublisher);
			break;
		case communication::InstrumentationId::occupancy:
			device::occupancy::instrumentKernel(context, is_exit, eventId, params, measurementsPublisher);
			break;
		case communication::InstrumentationId::gmem_access_coalescence:
			device::gmem_access_coalescence::instrumentKernel(context, is_exit, params, measurementsPublisher);
			break;
		case communication::InstrumentationId::branch_divergence:
			device::branch_divergence::instrumentKernel(context, is_exit, params, measurementsPublisher);
			break;
		default:
			break;
		}
	}
}

void nvbit_at_init() {
	logging::info("NVBit runtime initializing, version: {}", NVBIT_VERSION);

	// Make sure all managed variables are allocated on GPU
	setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);

	runtimeConfigPoller.initialize(config::get().runtime_config_path, config::get().runtime_config_polling_interval);
	measurementsPublisher.initialize(config::get().measurements_output_dir);
}

void nvbit_at_cuda_event(
		hipCtx_t context, int is_exit, nvbit_api_cuda_t eventId, const char* /* name */, void* params, hipError_t* /* pStatus */) {
	const auto launchEvents = {
			API_CUDA_cuLaunch, API_CUDA_cuLaunchKernel_ptsz, API_CUDA_cuLaunchGrid, API_CUDA_cuLaunchGridAsync, API_CUDA_cuLaunchKernel};
	if (boost::algorithm::any_of_equal(launchEvents, eventId)) {
		const auto instrumentationFunctions = runtimeConfigPoller.getConfig().getInstrumentationFunctions();
		instrumentKernelLaunch(context, is_exit, eventId, static_cast<cuLaunch_params*>(params), instrumentationFunctions);
	}
}

void nvbit_at_term() {
	logging::info("NVBit runtime exiting");
}