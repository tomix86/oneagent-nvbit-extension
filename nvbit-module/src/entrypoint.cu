#include "Configuration.h"
#include "Logger.h"
#include "device_functions/functions_registry.h"
#include "device_functions/count_instrs.h"
#include "device_functions/occupancy.h"
#include "communication/RuntimeConfigurationPoller.h"
#include "communication/MeasurementsPublisher.h"

#include <nvbit_tool.h> // Must be included only once!

#include <boost/algorithm/cxx11/any_of.hpp>

communication::RuntimeConfigurationPoller runtimeConfigPoller;
communication::MeasurementsPublisher measurementsPublisher;

static void instrumentKernelLaunch(hipCtx_t context, int is_exit, nvbit_api_cuda_t eventId, cuLaunch_params* params, const std::vector<std::string> & instrumentationFunctions) {
    for(const auto& functionName : instrumentationFunctions) {
        if(functionName == NAME_OF(INSTRUMENTATION__INSTRUCTIONS_COUNT)) {
            count_instr::instrumentKernelWithInstructionCounter(context, is_exit, eventId, params, measurementsPublisher);
        } else if (functionName == NAME_OF(INSTRUMENTATION__OCCUPANCY)) {
            occupancy::instrumentKernelWithOccupancyCounter(context, is_exit, eventId, params, measurementsPublisher);
        } else {
            logging::warning("Unexpected instrumentation function name", functionName);
        }
    }
}

void nvbit_at_init() {
    logging::info("NVBit runtime initializing");

    // Make sure all managed variables are allocated on GPU
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);

    runtimeConfigPoller.initialize(config::get().runtime_config_path, config::get().runtime_config_polling_interval);
    measurementsPublisher.initialize(config::get().measurements_output_dir);
}

void nvbit_at_cuda_event(hipCtx_t context, int is_exit, nvbit_api_cuda_t eventId, const char* /* name */, void* params, hipError_t* /* pStatus */) {
    const auto launchEvents = {API_CUDA_cuLaunch, API_CUDA_cuLaunchKernel_ptsz, API_CUDA_cuLaunchGrid, API_CUDA_cuLaunchGridAsync, API_CUDA_cuLaunchKernel};
    if (boost::algorithm::any_of_equal(launchEvents, eventId)) {
        const auto instrumentationFunctions = runtimeConfigPoller.getConfig().getInstrumentationFunctions();
        instrumentKernelLaunch(context, is_exit, eventId, static_cast<cuLaunch_params*>(params), instrumentationFunctions);
    }
}

void nvbit_at_term() {
    logging::info("NVBit runtime exiting");
}